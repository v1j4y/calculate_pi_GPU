#include "hip/hip_runtime.h"
/*
 * Author:  Vijay Gopal Chilkuri
 * Email:   vijay.gopal.c@gmail.com
 * Date:    12-08-2020
 */

#include "parallel_reduction.h"

// 2^13
#define NMC  8192 
#define LenVec 1024

// ----------------------------------------------------
// Parallel reduction on GPU based on presentation
// by Mark Harris, NVIDIA.
// 
// 
// reduction kernel level 0
//
// ----------------------------------------------------
__global__  void vectorReduction0(Vector g_idata, Vector g_odata){

    // Size automatically determined using third execution control parameter
    // when kernel is invoked.
    extern __shared__ float sdata[];

    int tid     = threadIdx.x;
    int index   = blockIdx.x * blockDim.x + threadIdx.x;

    // This instruction copies data from 
    // global to shared memory of each block.
    // Only threads of a block can access this shared memory.
    sdata[tid]  = g_idata.elements[index];

    // Synchronize threads, basically a barrier.
    __syncthreads();
    
    // Do the reduction in shared memory buffer
    // Thread Id:  0 - 1 - 2 - 3 - 4 - 5
    //             |  /    |  /    |  /
    //             0       2       4
    for(unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if(tid % (2*s) == 0)
        {
           sdata[tid] += sdata[tid + s];
        }
    }

    __syncthreads();

    // Write back result to global memory
    if(tid == 0) g_odata.elements[blockIdx.x] = sdata[0];
}

int parallel_reduction(void) 
{
    int i,j;
    // Allocate and initialize the matrices
    Vector  V     = AllocateVector(LenVec);

    // Timing stuff
    struct timeval t1, t2;
    double time = 0.0;
 
    // Initialize Matrix of grid points
    for(unsigned int i = 0; i < V.length; i++)
    {
        V.elements[i] = (LenVec/2 - i)*(LenVec/2 - i);
    }
 
    gettimeofday(&t1, 0);
    
    // Serial Reduction of Vector elements
    float sum = 0;

    for(unsigned int i=0; i < V.length; i++)
    {
        sum += V.elements[i];
    }
    printf("Serial Sum=%5.1f\n",sum);
    
    gettimeofday(&t2, 0);
    
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    
    printf("Time serial sum:  %3.1f ms \n", time);


    gettimeofday(&t1, 0);

    // Parallel reduction 
    int NBlocks;
    int NThreadsPerBlock;
    Vector Vout;
    Vector Vinp_d, Vout_d;
    int dimVec  = LenVec;
    int NBdim   = 128;
    int dimOutVec = dimVec/NBdim;
    Vout = AllocateZeroVector(dimOutVec);

    for(unsigned int i = 1; i < 2; i *= 2)
    {

        NBlocks           = dimVec/NBdim;
        NThreadsPerBlock  = NBdim;

        dim3 dimBlock(NThreadsPerBlock);
        dim3 dimGrid(NBlocks);

        // Create device vectors
        Vinp_d     = AllocateDeviceVector(V);
        Vout_d     = AllocateDeviceVector(Vout);

        // Copy data to device vector
        CopyToDeviceVector(Vinp_d, V);

        // Copy vectors to device

        printf("NBlocks = %d NThreadsPerBlock=%d \n",NBlocks,NThreadsPerBlock);

        vectorReduction0<<<dimGrid, dimBlock, NBlocks>>>(Vinp_d, Vout_d);

    }

    // Copy data from device
    CopyFromDeviceVector(Vout, Vout_d);

    printf("Output Vector\n");
    PrintVector(Vout.elements,Vout.length);

    sum = 0.0;
    for(unsigned int i = 0; i < Vout.length; i++)
      sum += Vout.elements[i];

    // print results
    printf("parallel Sum=%5.1f\n",sum);

    hipDeviceSynchronize();
    
    gettimeofday(&t2, 0);
    
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    
    printf("Time parallel sum:  %3.1f ms \n", time);


    // Free matrices
//  FreeMatrix(M);
    FreeVector(V);
    FreeVector(Vout);
    FreeDeviceVector(Vinp_d);
    FreeDeviceVector(Vout_d);

    return 0;
}
